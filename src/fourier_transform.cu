
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cmath>

__global__
void fourierTransform(float* dst, float* src, int width, int height) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;

  if (idx >= width || idy >= height) { return; }

  float real = 0.0;
  float imaginary = 0.0;
  for (int x = 0; x < width; x++) {
    for (int y = 0; y < height; y++) {
      real += src[y * width + x] * sinf((-2.0 * M_PI * idx * x / width) + (-2.0 * M_PI * idy * y / height));
      imaginary += src[y * width + x] * cosf((-2.0 * M_PI * idx * x / width) + (-2.0 * M_PI * idy * y / height));
    }
  }

  dst[idy * width + idx] = sqrtf((real * real) + (imaginary * imaginary));
}

extern "C" void fourierTransformWrapper(unsigned char* dst, unsigned char* src, int width, int height, int channels) {
  dim3 block(32, 32);
  dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

  float *h_dst = (float*)malloc(width*height*sizeof(float));
  float *h_src = (float*)malloc(width*height*sizeof(float));

  for (int x = 0; x < width * height * channels; x += channels) {
    h_src[x / channels] = src[x] / 255.0;
  }

  float *d_dst, *d_src;
  hipMalloc(&d_dst, width*height*sizeof(float));
  hipMalloc(&d_src, width*height*sizeof(float));
  hipMemcpy(d_src, h_src, width*height*sizeof(float), hipMemcpyHostToDevice);

  fourierTransform<<<block, grid>>>(d_dst, d_src, width, height);
  hipDeviceSynchronize();

  hipMemcpy(h_dst, d_dst, width*height*sizeof(float), hipMemcpyDeviceToHost);

  float max = 0.0;
  for (int x = 0; x < width * height; x++) {
    if (h_dst[x] > max) {
      max = h_dst[x];
    }
  }

  float c = 255.0 / log(1 + fabs(max));
  for (int x = 0; x < width * height; x++) {
    h_dst[x] = c * log(1 + fabs(h_dst[x]));

    dst[x * channels] = h_dst[x];
    dst[x * channels + 1] = h_dst[x];
    dst[x * channels + 2] = h_dst[x];
    dst[x * channels + 3] = 255;
  }

  hipFree(d_dst);
  hipFree(d_src);
  free(h_dst);
  free(h_src);
}