
#include <hip/hip_runtime.h>
__global__
void fourierTransform(unsigned char* dst, unsigned char* src, int width, int height) {
  int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
  int idy = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (idx >= width || idy >= height) { return; }
}

extern "C" void fourierTransformWrapper(unsigned char* dst, unsigned char* src, int width, int height) {
  dim3 block(32, 32);
  dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

  unsigned char *d_dst, *d_src;
  hipMalloc(&d_dst, width*height*sizeof(unsigned char));
  hipMalloc(&d_src, width*height*sizeof(unsigned char));
  hipMemcpy(d_src, src, width*height*sizeof(unsigned char), hipMemcpyHostToDevice);

  fourierTransform<<<block, grid>>>(d_dst, d_src, width, height);
  hipDeviceSynchronize();

  hipMemcpy(dst, d_dst, width*height*sizeof(unsigned char), hipMemcpyDeviceToHost);

  hipFree(d_dst);
  hipFree(d_src);
}