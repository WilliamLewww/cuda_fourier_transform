
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <stdio.h>
#include <complex.h>

hipFloatComplex cuexpf(hipFloatComplex value) {
  float exponent = expf(value.x);
  float real, imaginary;
  sincosf(value.y, &imaginary, &real);

  return make_hipFloatComplex(real * exponent, imaginary * exponent);
}

__global__
void recursiveFastFourierTransform(hipFloatComplex* dst, hipFloatComplex* src, int width, int height) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
}

extern "C" void fastFourierTransformWrapper2D(float* dst, float* src, int width, int height) {
  dim3 block(32, 32);
  dim3 grid((block.x + width - 1) / block.x, (block.y + height - 1) / block.y);

  hipFloatComplex* h_buffer = (hipFloatComplex*)malloc(width*height*sizeof(hipFloatComplex));
  for (int x = 0; x < width * height; x++) { h_buffer[x] = make_hipFloatComplex(src[x], 0.0); }

  hipFloatComplex* d_buffer;
  hipMalloc(&d_buffer, width*height*sizeof(hipFloatComplex));
  hipMemcpy(d_buffer, h_buffer, width*height*sizeof(hipFloatComplex), hipMemcpyHostToDevice);

  hipFloatComplex* d_bufferResult;
  hipMalloc(&d_bufferResult, width*height*sizeof(hipFloatComplex));

  recursiveFastFourierTransform<<<grid, block>>>(d_bufferResult, d_buffer, width, height);

  hipDeviceReset();
}