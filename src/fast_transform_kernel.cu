
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <stdio.h>
#include <complex.h>

hipFloatComplex cuexpf(hipFloatComplex value) {
  float exponent = expf(value.x);
  float real, imaginary;
  sincosf(value.y, &imaginary, &real);

  return make_hipFloatComplex(real * exponent, imaginary * exponent);
}

extern "C" void fastFourierTransformWrapper2D(float* dst, float* src, int width, int height) {
  
}