
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cmath>
#include <complex>

void recursiveFastFourierTransformCPU(std::complex<double>* bufferCombine, std::complex<double>* bufferSplit, int size, int stride) {
  if (stride < size) {
    recursiveFastFourierTransformCPU(bufferSplit, bufferCombine, size, stride * 2);
    recursiveFastFourierTransformCPU(bufferSplit + stride, bufferCombine + stride, size, stride * 2);
 
    for (int i = 0; i < size; i += 2 * stride) {
      std::complex<double> t = std::exp(-std::complex<double>(0, 1) * M_PI * double(i) / double(size)) * bufferSplit[i + stride];
      bufferCombine[i / 2] = bufferSplit[i] + t;
      bufferCombine[(i + size)/2] = bufferSplit[i] - t;
    }
  }
}

void fastFourierTransformCPU(float* dst, float* src, int width, int height) {
  // std::complex<double> buffer[] = {1, 1, 1, 1, 0, 0, 0, 0};
  // std::complex<double> bufferClone[] = {1, 1, 1, 1, 0, 0, 0, 0};

  std::complex<double>* buffer = (std::complex<double>*)malloc(width*sizeof(std::complex<double>));
  for (int x = 0; x < width; x++) { buffer[x] = src[x]; }

  std::complex<double>* bufferClone = (std::complex<double>*)malloc(width*sizeof(std::complex<double>));
  memcpy(bufferClone, buffer, width*sizeof(std::complex<double>));

  recursiveFastFourierTransformCPU(buffer, bufferClone, width, 1);

  for (int x = 0; x < width; x++) {
    printf("%f\n", sqrtf((buffer[x].real() * buffer[x].real()) + (buffer[x].imag() * buffer[x].imag())));
  }
}