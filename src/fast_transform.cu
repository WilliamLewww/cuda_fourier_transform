
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cmath>
#include <complex>

void recursiveFastFourierTransformCPU(std::complex<float>* bufferCombine, std::complex<float>* bufferSplit, int size, int stride) {
  if (stride < size) {
    recursiveFastFourierTransformCPU(bufferSplit, bufferCombine, size, stride * 2);
    recursiveFastFourierTransformCPU(bufferSplit + stride, bufferCombine + stride, size, stride * 2);
 
    for (int i = 0; i < size; i += 2 * stride) {
      std::complex<float> t = std::exp(-std::complex<float>(0, 1) * float(M_PI) * float(i) / float(size)) * bufferSplit[i + stride];
      bufferCombine[i / 2] = bufferSplit[i] + t;
      bufferCombine[(i + size)/2] = bufferSplit[i] - t;
    }
  }
}

void fastFourierTransformCPU(float* dst, float* src, int width, int height) {
  std::complex<float>* imageBuffer = (std::complex<float>*)malloc(width*height*sizeof(std::complex<float>));

  std::complex<float>* buffer = (std::complex<float>*)malloc(width*sizeof(std::complex<float>));
  std::complex<float>* bufferClone = (std::complex<float>*)malloc(width*sizeof(std::complex<float>));

  for (int y = 0; y < height; y++) {
    for (int x = 0; x < width; x++) { 
      buffer[x] = src[y * width + x]; 
    }
    memcpy(bufferClone, buffer, width*sizeof(std::complex<float>));

    recursiveFastFourierTransformCPU(buffer, bufferClone, width, 1);

    for (int x = 0; x < width; x++) {
      imageBuffer[x * height + y] = buffer[x];
    }
  }

  for (int y = 0; y < height; y++) {
    memcpy(buffer, &imageBuffer[y * width], width*sizeof(std::complex<float>));
    memcpy(bufferClone, buffer, width*sizeof(std::complex<float>));

    recursiveFastFourierTransformCPU(buffer, bufferClone, width, 1);

    for (int x = 0; x < width; x++) {
      imageBuffer[y * width + x] = buffer[x];
    }
  }

  for (int x = 0; x < width * height; x++) {
    dst[x] = sqrtf((imageBuffer[x].real() * imageBuffer[x].real()) + (imageBuffer[x].imag() * imageBuffer[x].imag()));
  }
}