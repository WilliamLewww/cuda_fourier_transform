
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cmath>

float* recursiveFourierTransformCPU(float* samples, int size) {
  if (size == 1) {
    return samples;
  }

  float* even = (float*)malloc(size/2*sizeof(float));
  float* odd = (float*)malloc(size/2*sizeof(float));

  for (int x = 0; x < size / 2; x++) {
    even[x] = samples[2 * x];
    odd[x] = samples[2  * x + 1];
  }

  float* fEven = recursiveFourierTransformCPU(even, size / 2);
  float* fOdd = recursiveFourierTransformCPU(odd, size / 2);

  float* bins = (float*)malloc(size*sizeof(float));
  for (int x = 0; x < size / 2; x++) {
    float real = sinf(-2.0 * M_PI * x / size) * fOdd[x];
    float imaginary = cosf(-2.0 * M_PI * x / size) * fOdd[x];

    bins[x] = fEven[x] + sqrtf((real * real) + (imaginary * imaginary));
    bins[x + (size / 2)] = fEven[x] - sqrtf((real * real) + (imaginary * imaginary));
  }

  return bins;
}

extern "C" void fastFourierTransformCPU(float* dst, float* src, int width, int height) {
  for (int row = 0; row < height; row++) {
    recursiveFourierTransformCPU(&src[row * width], width);
  }
}